#include "hip/hip_runtime.h"
#include "caffe/layer.hpp"

#include "caffe/vision_layers.hpp"

namespace caffe {
/**
 * GPU implementation of the forward pass of image cropping.
 *
 * @param nthreads the number of threads overall (should be equal to top[0]->count()
 * @param bottom_data pointer to the GPU data of the bottom layer
 * TODO(ssafar): to be done.
 */
template <typename Dtype>
__global__ void CropForward(const int nthreads,
    const Dtype* bottom_data,
    const int bottom_n, const int bottom_c, const int bottom_h,
    const int bottom_w,
    Dtype* const top_data,
    const int valid_h_begin, const int top_h,
    const int valid_w_begin, const int top_w) {

  const int top_c = bottom_c;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int top_pos_w = index % top_w;
    const int top_pos_h = index / top_w % top_h;
    const int pos_c = index / top_w / top_h % top_c;
    const int pos_n = index / top_w / top_h / top_c;


    const int bottom_pos_w = top_pos_w + valid_w_begin;
    const int bottom_pos_h = top_pos_h + valid_h_begin;

    Dtype* const top_addr = top_data + index;

    int bottom_offset = pos_n;
    bottom_offset = bottom_offset * bottom_c + pos_c;
    bottom_offset = bottom_offset * bottom_h + bottom_pos_h;
    bottom_offset = bottom_offset * bottom_w + bottom_pos_w;
    const Dtype* bottom_addr = bottom_data + bottom_offset;

    *top_addr = *bottom_addr;
  }
}

template <typename Dtype>
__global__ void CropBackward(const int nthreads,
    Dtype* const bottom_diff,
    const int bottom_n, const int bottom_c, const int bottom_h,
    const int bottom_w,
    const Dtype* top_diff,
    const int valid_h_begin, const int top_h,
    const int valid_w_begin, const int top_w) {
  const int top_c = bottom_c;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int bottom_pos_w = index % bottom_w;
    const int bottom_pos_h = index / bottom_w % bottom_h;
    const int pos_c = index / bottom_w / bottom_h % bottom_c;
    const int pos_n = index / bottom_w / bottom_h / bottom_c;

    const int top_pos_h = bottom_pos_h - valid_h_begin;
    const int top_pos_w = bottom_pos_w - valid_w_begin;

    Dtype* const bottom_addr = bottom_diff + index;

    if (top_pos_w >= 0 && top_pos_w < top_w &&
        top_pos_h >= 0 && top_pos_h < top_h) {

      int top_offset = pos_n;
      top_offset = top_offset * top_c + pos_c;
      top_offset = top_offset * top_h + top_pos_h;
      top_offset = top_offset * top_w + top_pos_w;
      const Dtype* top_addr = top_diff + top_offset;

      *bottom_addr = *top_addr;
    } else {
      *bottom_addr = 0;
    }
  }
}

template <typename Dtype>
void CropLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  // One thread for each output (top) value.
  int count = top[0]->count();

  CropForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, bottom_data,
    bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(),
    top_data,
    valid_h_begin, top[0]->height(), valid_w_begin, top[0]->width());
}

template <typename Dtype>
void CropLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  Dtype* const bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* top_diff = top[0]->gpu_diff();

  // One thread for each output (bottom) value.
  int count = bottom[0]->count();

  CropBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, bottom_diff,
    bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(),
    top_diff,
    valid_h_begin, top[0]->height(), valid_w_begin, top[0]->width());
}

INSTANTIATE_LAYER_GPU_FUNCS(CropLayer);

}  // namespace caffe
