#include "hip/hip_runtime.h"
#include "caffe/layer.hpp"

#include "caffe/vision_layers.hpp"

namespace caffe {

inline __device__ int offset_4d(int n, int c, int h, int w,
    int size_c, int size_h, int size_w) {
  return ((((n * size_c) + c) * size_h) + h) * size_w + w;
}

template <typename Dtype>
__global__ void UnpoolForward(const int nthreads,
    const Dtype* const bottom_data,
    const int bottom_n, const int bottom_c, const int bottom_h,
    const int bottom_w,
    Dtype* const top_data,
    const int kernel_w, const int kernel_h)

{
  const int top_c = bottom_c;
  const int top_w = bottom_w * kernel_w;
  const int top_h = bottom_h * kernel_h;

  CUDA_KERNEL_LOOP(index, nthreads) {
    // Go over the top layer, do the same thing as we did before.
    const int top_pos_w = index % top_w;
    const int top_pos_h = index / top_w % top_h;
    const int pos_c = index / top_w / top_h % top_c;
    const int pos_n = index / top_w / top_h / top_c;

    const int bottom_pos_w = top_pos_w / kernel_w;
    const int bottom_pos_h = top_pos_h / kernel_h;

    Dtype* const top_addr = top_data + index;

    // int bottom_offset = pos_n;
    // bottom_offset = bottom_offset * bottom_c + pos_c;
    // bottom_offset = bottom_offset * bottom_h + bottom_pos_h;
    // bottom_offset = bottom_offset * bottom_w + bottom_pos_w;
    const int bottom_offset = offset_4d(pos_n, pos_c, bottom_pos_h, bottom_pos_w,
        bottom_c, bottom_h, bottom_w);

    const Dtype* bottom_addr = bottom_data + bottom_offset;

    *top_addr = *bottom_addr;
  }
}

template <typename Dtype>
__global__ void UnpoolBackward(const int nthreads,
    Dtype* const bottom_diff,
    const int bottom_n, const int bottom_c, const int bottom_h,
    const int bottom_w,
    const Dtype* top_diff,
    const int kernel_w, const int kernel_h) {
  // Go along the bottom, collecting individual components using a loop. Not
  // super efficient, but at least works.
  const int top_c = bottom_c;
  const int top_h = bottom_h * kernel_h;
  const int top_w = bottom_w * kernel_w;

  CUDA_KERNEL_LOOP(index, nthreads) {
    // Go over the top layer, do the same thing as we did before.
    const int bottom_pos_w = index % bottom_w;
    const int bottom_pos_h = index / bottom_w % bottom_h;
    const int pos_c = index / bottom_w / bottom_h % bottom_c;
    const int pos_n = index / bottom_w / bottom_h / bottom_c;

    const int top_base_pos_h = bottom_pos_h * kernel_h;
    const int top_base_pos_w = bottom_pos_w * kernel_w;

    const Dtype* const top_base_addr = top_diff + offset_4d(pos_n, pos_c,
        top_base_pos_h, top_base_pos_w,
        top_c, top_h, top_w);

    Dtype* const bottom_addr = bottom_diff + offset_4d(pos_n, pos_c,
        bottom_pos_h, bottom_pos_w,
        bottom_c, bottom_h, bottom_w);

    *bottom_addr = 0;

    for (int delta_h = 0; delta_h < kernel_h; ++delta_h) {
      for (int delta_w = 0; delta_w < kernel_w; ++delta_w) {
        const Dtype* const top_addr = top_base_addr + top_w * delta_h + delta_w;
        *bottom_addr += *top_addr;
      }
    }
  }
}



template <typename Dtype>
void UnpoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  // One thread for each output (top) value.
  int count = top[0]->count();

  UnpoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, bottom_data,
    bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(),
    top_data,
    kernel_h_, kernel_w_);

}

template <typename Dtype>
void UnpoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  Dtype* const bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* top_diff = top[0]->gpu_diff();

  // One thread for each output (bottom) value.
  int count = bottom[0]->count();

  UnpoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, bottom_diff,
    bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width(),
    top_diff,
    kernel_h_, kernel_w_);
}

INSTANTIATE_LAYER_GPU_FUNCS(UnpoolingLayer);

}  // namespace caffe

